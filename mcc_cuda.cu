#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <chrono>

#define MAX(a,b) ((a) > (b) ? (a) : (b))
#define CHUNK_SIZE 4

// Device paired function
__device__ int paired_device(int a, int b) {
    // Replace this with your implementation of the paired function
    return a * b;  // Example implementation, replace with the actual logic
}

int ERT = 1;

// Host paired function
int paired_host(int a, int b) {
    // Replace this with your implementation of the paired function
    return a * b;  // Example implementation, replace with the actual logic
	
}

// Device max function
__device__ int max_device(int a, int b) {
    return MAX(a, b);
}


__global__ void computeQ(int N, int c1, int** d_Q1, int** d_Qbp1) {
    int globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int c3_start = globalThreadIdx * CHUNK_SIZE;
    int c3_end = c3_start + CHUNK_SIZE;
	
	int ERT=1;

    for (int c3 = c3_start; c3 < c3_end && c3 < (c1 + 1) / 2; c3++) {
        if (c3 >= max_device(0, -N + c1 + 1)) {
            d_Q1[N-c1+c3-1][N-c1+2*c3] = d_Q1[N-c1+c3-1][N-c1+2*c3-1];
            for (int c5 = 0; c5 <= c3; c5++) {
                d_Qbp1[c5+(N-c1+c3-1)][N-c1+2*c3] = d_Q1[c5+(N-c1+c3-1)+1][N-c1+2*c3-1] * ERT * paired_device(c5+(N-c1+c3-1),N-c1+2*c3-1);
                d_Q1[N-c1+c3-1][N-c1+2*c3] += d_Q1[N-c1+c3-1][c5+(N-c1+c3-1)] * d_Qbp1[c5+(N-c1+c3-1)][N-c1+2*c3];
            }
        }
    }
}

int main() {
    int N = 5000;  // Example size
    int **h_Q1, **d_Q1, **h_Qbp1, **d_Qbp1, **cpu_Q1, **cpu_Qbp1;
    int *d_Q1_data, *d_Qbp1_data;

    // Allocate and initialize host memory
    h_Q1 = (int**)malloc(N * sizeof(int*));
    h_Qbp1 = (int**)malloc(N * sizeof(int*));
    cpu_Q1 = (int**)malloc(N * sizeof(int*));
    cpu_Qbp1 = (int**)malloc(N * sizeof(int*));

    for (int i = 0; i < N; i++) {
        h_Q1[i] = (int*)malloc(N * sizeof(int));
        h_Qbp1[i] = (int*)malloc(N * sizeof(int));
        cpu_Q1[i] = (int*)malloc(N * sizeof(int));
        cpu_Qbp1[i] = (int*)malloc(N * sizeof(int));

        for (int j = 0; j < N; j++) {
            h_Q1[i][j] = rand() % 100;  // Example initialization
            h_Qbp1[i][j] = rand() % 100;  // Example initialization
            cpu_Q1[i][j] = h_Q1[i][j];
            cpu_Qbp1[i][j] = h_Qbp1[i][j];
        }
    }

    // Allocate device memory
    hipMalloc(&d_Q1_data, N * N * sizeof(int));
    hipMalloc(&d_Qbp1_data, N * N * sizeof(int));
    hipMalloc(&d_Q1, N * sizeof(int*));
    hipMalloc(&d_Qbp1, N * sizeof(int*));

    int **h_Q1_array = (int **)malloc(N * sizeof(int *));
    int **h_Qbp1_array = (int **)malloc(N * sizeof(int *));

    for (int i = 0; i < N; i++) {
        h_Q1_array[i] = d_Q1_data + i * N;
        h_Qbp1_array[i] = d_Qbp1_data + i * N;
    }

    hipMemcpy(d_Q1, h_Q1_array, N * sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(d_Qbp1, h_Qbp1_array, N * sizeof(int *), hipMemcpyHostToDevice);

    // Copy data to device
    for (int i = 0; i < N; i++) {
        hipMemcpy(h_Q1_array[i], h_Q1[i], N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(h_Qbp1_array[i], h_Qbp1[i], N * sizeof(int), hipMemcpyHostToDevice);
    }

    // GPU computation
    int threadsPerBlock = 256;
    int numBlocks = (N / (threadsPerBlock * CHUNK_SIZE)) + 1;

    auto gpu_start = std::chrono::high_resolution_clock::now();

    for (int c1 = 1; c1 < 2 * N - 2; c1 += 1) {
        computeQ<<<numBlocks, threadsPerBlock>>>(N, c1, d_Q1, d_Qbp1);
        hipDeviceSynchronize();
    }

    auto gpu_end = std::chrono::high_resolution_clock::now();

    // Copy results back to host
    for (int i = 0; i < N; i++) {
        hipMemcpy(h_Q1[i], h_Q1_array[i], N * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_Qbp1[i], h_Qbp1_array[i], N * sizeof(int), hipMemcpyDeviceToHost);
    }

    // CPU computation
    auto cpu_start = std::chrono::high_resolution_clock::now();

    for (int c1 = 1; c1 < 2 * N - 2; c1 += 1) {
        for (int c3 = max(0, -N + c1 + 1); c3 < (c1 + 1) / 2; c3++) {
            cpu_Q1[N-c1+c3-1][N-c1+2*c3] = cpu_Q1[N-c1+c3-1][N-c1+2*c3-1];
            for (int c5 = 0; c5 <= c3; c5++) {
                cpu_Qbp1[c5+(N-c1+c3-1)][N-c1+2*c3] = cpu_Q1[c5+(N-c1+c3-1)+1][N-c1+2*c3-1] * ERT * paired_host(c5+(N-c1+c3-1),N-c1+2*c3-1);
                cpu_Q1[N-c1+c3-1][N-c1+2*c3] += cpu_Q1[N-c1+c3-1][c5+(N-c1+c3-1)] * cpu_Qbp1[c5+(N-c1+c3-1)][N-c1+2*c3];
            }
        }
    }

    auto cpu_end = std::chrono::high_resolution_clock::now();

    // Validate results
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            assert(h_Q1[i][j] == cpu_Q1[i][j]);
            assert(h_Qbp1[i][j] == cpu_Qbp1[i][j]);
        }
    }

    // Print timings
    std::chrono::duration<double, std::milli> gpu_duration = gpu_end - gpu_start;
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
    printf("GPU calculation took: %f ms\n", gpu_duration.count());
    printf("CPU calculation took: %f ms\n", cpu_duration.count());

    // Free device memory
    hipFree(d_Q1_data);
    hipFree(d_Qbp1_data);
    hipFree(d_Q1);
    hipFree(d_Qbp1);

    // Free host memory
    for (int i = 0; i < N; i++) {
        free(h_Q1[i]);
        free(h_Qbp1[i]);
        free(cpu_Q1[i]);
        free(cpu_Qbp1[i]);
    }
    free(h_Q1);
    free(h_Qbp1);
    free(cpu_Q1);
    free(cpu_Qbp1);
    free(h_Q1_array);
    free(h_Qbp1_array);

    return 0;
}

