#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <chrono>

#define MIN(a,b) ((a) < (b) ? (a) : (b))

__global__ void computeCK(int n, int** d_ck, int** d_w) {
    int h0 = blockIdx.x * blockDim.x + threadIdx.x;

    if (h0 < 0) {
        for (int i2 = -h0 + 1; i2 < n - h0; i2++) {
            d_ck[-h0][n - h0] = MIN(d_ck[-h0][n - h0], (d_w[-h0][n - h0] + d_ck[-h0][i2]) + d_ck[i2][n - h0]);
        }
    }
}

// Host implementation for CK computation
void computeCKHost(int n, int** h_ck, int** h_w) {
    for (int w0 = 2; w0 < n; w0++) {
        for (int h0 = -n + w0; h0 < 0; h0++) {
            for (int i2 = -h0 + 1; i2 < w0 - h0; i2++) {
                h_ck[-h0][w0 - h0] = MIN(h_ck[-h0][w0 - h0], (h_w[-h0][w0 - h0] + h_ck[-h0][i2]) + h_ck[i2][w0 - h0]);
            }
        }
    }
}

int main() {
    int n = 1000;  // Example size
    int **h_ck, **d_ck, **h_w, **d_w;
    int *d_ck_data, *d_w_data;

    // Allocate and initialize host memory
    h_ck = (int**)malloc(n * sizeof(int*));
    h_w = (int**)malloc(n * sizeof(int*));

    for (int i = 0; i < n; i++) {
        h_ck[i] = (int*)malloc(n * sizeof(int));
        h_w[i] = (int*)malloc(n * sizeof(int));

        for (int j = 0; j < n; j++) {
            h_ck[i][j] = INT_MAX;  // Initialize to a large value
            h_w[i][j] = rand() % 100;  // Example initialization
        }
    }

    // Allocate device memory
    hipMalloc(&d_ck_data, n * n * sizeof(int));
    hipMalloc(&d_ck, n * sizeof(int*));
    hipMalloc(&d_w_data, n * n * sizeof(int));
    hipMalloc(&d_w, n * sizeof(int*));

    int **h_ck_array = (int **)malloc(n * sizeof(int *));
    int **h_w_array = (int **)malloc(n * sizeof(int *));

    for (int i = 0; i < n; i++) {
        h_ck_array[i] = d_ck_data + i * n;
        h_w_array[i] = d_w_data + i * n;
    }

    hipMemcpy(d_ck, h_ck_array, n * sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w_array, n * sizeof(int *), hipMemcpyHostToDevice);

    // Copy data to device
    for (int i = 0; i < n; i++) {
        hipMemcpy(h_ck_array[i], h_ck[i], n * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(h_w_array[i], h_w[i], n * sizeof(int), hipMemcpyHostToDevice);
    }

    // GPU computation
    int threadsPerBlock = 256;
    int numBlocks = (n / threadsPerBlock) + 1;

    auto gpu_start = std::chrono::high_resolution_clock::now();

    for (int w0 = 2; w0 < n; w0++) {
        computeCK<<<numBlocks, threadsPerBlock>>>(n, d_ck, d_w);
        hipDeviceSynchronize();
    }

    auto gpu_end = std::chrono::high_resolution_clock::now();

    // Copy results back to host
    for (int i = 0; i < n; i++) {
        hipMemcpy(h_ck[i], h_ck_array[i], n * sizeof(int), hipMemcpyDeviceToHost);
    }

    // Host computation
    auto cpu_start = std::chrono::high_resolution_clock::now();

    computeCKHost(n, h_ck, h_w);

    auto cpu_end = std::chrono::high_resolution_clock::now();


// Validate results
for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
        assert(h_ck[i][j] == h_ck_array[i][j]);
    }
}

    // Print timings
    std::chrono::duration<double, std::milli> gpu_duration = gpu_end - gpu_start;
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;
    printf("GPU calculation took: %f ms\n", gpu_duration.count());
    printf("CPU calculation took: %f ms\n", cpu_duration.count());

    // Free device memory
    hipFree(d_ck_data);
    hipFree(d_ck);
    hipFree(d_w_data);
    hipFree(d_w);

    // Free host memory
    for (int i = 0; i < n; i++) {
        free(h_ck[i]);
        free(h_w[i]);
    }
    free(h_ck);
    free(h_w);
    free(h_ck_array);
    free(h_w_array);

    return 0;
}
